
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

/* Every thread gets exactly one value in the unsorted array. */
#define THREADS 128 // 2^7
#define BLOCKS 1024 // 2^10
#define NUM_VALS THREADS*BLOCKS

void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %.3fs\n", elapsed);
}

__global__ void bitonic_sort_step(int *dev_values, int j, int k)
{
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      if (dev_values[i]>dev_values[ixj]) {
        // swap
        int temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (dev_values[i]<dev_values[ixj]) {
        // swap
        int temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
  }
}

void bitonic_sort(int *values)
{
  int *dev_values;
  size_t size = NUM_VALS * sizeof(int);

  hipMalloc((void**) &dev_values, size);
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

  dim3 blocks(BLOCKS,1);    
  dim3 threads(THREADS,1);  

  int j, k;
  /* Major step */
  for (k = 2; k <= NUM_VALS; k <<= 1) {
    /* Minor step */
    for (j=k>>1; j>0; j=j>>1) {
      bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
    }
  }
  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
  hipFree(dev_values);
}

int main(int argc, char const *argv[])
{
  clock_t start, stop;

  int *values = (int*)malloc(NUM_VALS * sizeof(int));

  FILE *f = fopen("reverse_dataset.txt", "r");

  for(int i=0;i< NUM_VALS; i++) {
    fscanf(f, "%d\n", &values[i]);
  }

  printf("Hello\n");
  start  = clock();
  bitonic_sort(values);
  for(int i=0; i < 20 ;i ++) {
    printf("%d\n", values[i]);
  }
  stop = clock();

  print_elapsed(start, stop);
  return 0;
}
