
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>



#define THREADS 128 // 2^7
#define BLOCKS 1024 // 2^10
#define NUM_VALS THREADS*BLOCKS

#define checkCudaErrors(func)                                                \
{                                                                       \
  hipError_t E  = func;                                                \
  if(E != hipSuccess)                                                  \
  {                                                                     \
    printf( "\nError at line: %d ", __LINE__);                          \
    printf( "\nError:  %s ", hipGetErrorString(E));                    \
  }                                                                     \
}  

void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %.3fs\n", elapsed);
}

// GPU helper function
// calculate the id of the current thread
__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

//
// Perform a full mergesort on our section of the data.
//
__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}
__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    long start = width*idx*slices, 
         middle, 
         end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}


//
// Finally, sort something
// gets called by gpu_mergesort() for each slice
//

void mergesort(long* data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {

    //
    // Allocate two arrays on the GPU
    // we switch back and forth between them during the sort
    //
    long* D_data;
    long* D_swp;
    dim3* D_threads;
    dim3* D_blocks;
    
    // Actually allocate the two arrays
    checkCudaErrors(hipMalloc((void**) &D_data, size * sizeof(long)));
    checkCudaErrors(hipMalloc((void**) &D_swp, size * sizeof(long)));


    // Copy from our input list into the first array
    checkCudaErrors(hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice));

    //
    // Copy the thread / block info to the GPU as well
    //
    checkCudaErrors(hipMalloc((void**) &D_threads, sizeof(dim3)));
    checkCudaErrors(hipMalloc((void**) &D_blocks, sizeof(dim3)));

 
    checkCudaErrors(hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice));



    long* A = D_data;
    long* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;


    for (int width = 2; width < (size << 1); width <<= 1) {
        long slices = size / ((nThreads) * width) + 1;


        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }


    checkCudaErrors(hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost));

    
    checkCudaErrors(hipFree(A));
    checkCudaErrors(hipFree(B));
   
}




int main(int argc, char const *argv[])
{
  clock_t start, stop;

  long *values = (long*)malloc(NUM_VALS * sizeof(long));

  FILE *f = fopen("reverse_dataset.txt", "r");

  for(int i=0;i< NUM_VALS; i++) {
    fscanf(f, "%d\n", &values[i]);
  }

  dim3 blocks(BLOCKS,1);    
  dim3 threads(THREADS,1); 
  start  = clock();
  mergesort(values, NUM_VALS, threads, blocks);
  stop = clock();

  print_elapsed(start, stop);
  return 0;
}
